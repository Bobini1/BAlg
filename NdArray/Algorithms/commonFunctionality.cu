//
// Created by bobini on 07.01.2022.
//

#include "commonFunctionality.cuh"

void BAlg::Algorithms::Implementations::checkErrors()
{
    hipError_t cudaStatus;
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error(std::string("hipDeviceSynchronize returned error code after launching addKernel: \n") + hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error(std::string("hipDeviceSynchronize returned error code after launching addKernel: \n") + hipGetErrorString(cudaStatus));
    }
}

hipDeviceProp_t BAlg::Algorithms::Implementations::getDeviceProperties() {
    static hipDeviceProp_t props;
    static bool initialized = false;
    if (initialized) {
        return props;
    }
    int device;
    hipGetDevice(&device);
    auto error  = hipGetDeviceProperties(&props, device);
    if (error != hipSuccess) {
        throw std::runtime_error("Error getting device properties");
    }
    return props;
}