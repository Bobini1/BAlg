//
// Created by bobini on 07.01.2022.
//

#include "commonFunctionality.cuh"

void BAlg::Algorithms::Implementations::checkErrors()
{
    hipError_t cudaStatus;
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
}

hipDeviceProp_t BAlg::Algorithms::Implementations::getDeviceProperties() {
    static hipDeviceProp_t props;
    static bool initialized = false;
    if (initialized) {
        return props;
    }
    int device;
    hipGetDevice(&device);
    auto error  = hipGetDeviceProperties(&props, device);
    if (error != hipSuccess) {
        throw std::runtime_error("Error getting device properties");
    }
    return props;
}