﻿
#include "Algorithms.cuh"
#include <iostream>
#include <vector>

int main()
{
    auto* arr = new double[100000];
    for (int i = 0; i < 100000; i++)
    {
        arr[i] = i + 1;
    }
    
    std::cout << BAlg::Algorithms::reduce(arr, 100000, BAlg::Algorithms::Operation::ADD) << std::endl;

    return 0;
}